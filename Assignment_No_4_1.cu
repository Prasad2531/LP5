/*
 * Problem Statement :-
    Write a CUDA Program using CUDA C for :
        1. Addition of two large vectors
        2. Matrix Multiplication 
*/
//O/P Cmd: 1.) nvcc file.cu

//2.) /a.out

//1. Addition of two large vectors

#include <iostream> 
#include <hip/hip_runtime.h>

#define N 100000
#define THREADS_PER_BLOCK 1024

__global__ void add(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        c[i] = a[i] + b[i];
}

int main()
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate memory on host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Initialize arrays
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
        c[i] = 0;
    }

    // Allocate memory on device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy input data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel on device
    // grid represent => size of the grid of blocks that will be launched on the device(GPU)
    // 1st dimension => number of blocks required to launch N threads with THREADS_PER_BLOCK threads per block
    // 2nd and 3rd dimension => 1 since we are launching a one-dimensional grid
    dim3 grid((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1);
    // The block variable is used to specify the size of each block
    dim3 block(THREADS_PER_BLOCK, 1, 1);
    // Each thread in the grid will execute the kernel function
    add<<<grid, block>>>(d_a, d_b, d_c);

    // Copy output data from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Display the result
    for (int i = 0; i < N; i++)
    {
        printf("%d ", c[i]);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free memory on host
    free(a);
    free(b);
    free(c);

    return 0;
}